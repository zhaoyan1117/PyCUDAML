#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <string.h>
#include <iomanip>
#include <math.h>
#include <time.h>

#include "KMeans.cuh"

#define NUM_THREADS 10

inline int calc_num_blks(int value)
{
  return (value + NUM_THREADS - 1) / NUM_THREADS;
}

void kmeans(int k, const float **X,
            int n, int d,
            int max_iter, float threshold,
            float **cluster_centers, int* cluster_assignments)
{
  srand(time(NULL));

  /* Copy input to GPU as a flatten array */
  float *device_X;
  if (hipMalloc((void **) &device_X, n * d * sizeof(float)) != hipSuccess)
    throw;
  for (int i = 0; i < n; i++)
  {
    if (hipMemcpy(device_X+i*d, X[i], d * sizeof(float), hipMemcpyHostToDevice) != hipSuccess)
      throw;
  }

  /* Init device cluster centers as a flatten array */
  float *device_cluster_centers;
  if (hipMalloc((void **) &device_cluster_centers, k * d * sizeof(float)) != hipSuccess)
    throw;

  hiprandState* device_states;
  hipMalloc((void **) &device_states, k*sizeof(hiprandState));
  cu_init_cluster_centers <<< calc_num_blks(k), NUM_THREADS >>> \
        (k, (const float*) device_X, n, d, device_cluster_centers, \
         device_states, unsigned(time(NULL)));

  /* Init device cluster assignments */
  int *cluster_assignments;
  if (hipMalloc((void **) &cluster_assignments, n*sizeof(int)) != hipSuccess)
    throw;







  init_cluster_centers(k, X, n, d, cluster_centers);

  /* First round. */
  int delta;
  float delta_rate;
  int cur_iter = 0;

  delta = assign_clusters(k, X, n, d, cluster_assignments, (const float**) cluster_centers);
  calc_cluster_centers(k, X, n, d, (const int*) cluster_assignments, cluster_centers);
  delta_rate = ((float)delta)/((float)n);

  while (!is_terminated(cur_iter, max_iter, delta_rate, threshold))
  {
    std::cout << '\r' << '[' << cur_iter << "/" << max_iter << ']';
    std::cout.flush();

    delta = assign_clusters(k, X, n, d,
                            cluster_assignments, (const float**) cluster_centers);
    calc_cluster_centers(k, X, n, d, (const int*) cluster_assignments, cluster_centers);

    delta_rate = ((float)delta)/((float)n);
    cur_iter++;
  }

  /* Clean up. */
  hipFree(device_X);
  hipFree(device_states);
  hipFree(device_cluster_centers);
}

__global__ void cu_init_cluster_centers(int k, const float *device_X, int n, int d,
                                        float *device_cluster_centers,
                                        hiprandState *device_states, unsigned long seed)
{
  int my_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (my_id < k)
  {
    hiprandState my_state = device_states[my_id];
    hiprand_init(seed, my_id, 0, &my_state);
    int X_i;
    X_i = (int) (hiprand_uniform(&my_state) * n);
    memcpy(device_cluster_centers + my_id * d, device_X + X_i * d, sizeof(float) * d);
  }
}

bool is_terminated(int cur_iter, int max_iter, float delta_rate, float threshold)
{
  if (max_iter)
  {
    if (cur_iter > max_iter)
    {
      std::cout << '\r'
                << "Iteration: ["
                << cur_iter-1 << "/" << max_iter
                << "] | Delta rate: "
                << delta_rate
                << std::endl;
      return 1;
    }
    else if (delta_rate < threshold)
    {
      std::cout << '\r'
                << "Iteration: ["
                << cur_iter-1 << "/" << max_iter
                << "] | Delta rate: "
                << delta_rate
                << std::endl;
      return 1;
    }
    else
    {
      return 0;
    }
  }
  else
  {
    if (delta_rate < threshold)
    {
      std::cout << '\r'
                << "Iteration: ["
                << cur_iter-1 << "/" << max_iter
                << "] | Delta rate: "
                << delta_rate
                << std::endl;
      return 1;
    }
    else
    {
      return 0;
    }
  }
}

void init_cluster_centers(int k, const float **X, int n, int d, float **cluster_centers)
{
  for (int k_i = 0; k_i < k; k_i++)
  {
    int X_i = rand() % n;
    if (!(cluster_centers[k_i] = (float*)malloc(d*sizeof(float))))
    {
      throw;
    }
    memcpy(cluster_centers[k_i], X[X_i], d*sizeof(float));
  }
}

int assign_clusters(int k, const float **X, int n, int d,
          int *cluster_assignments, const float **cluster_centers)
{
  float cur_dist, best_dist;
  int best_cluster;
  int delta = 0;

  for (int X_i = 0; X_i < n; X_i++)
  {
    best_dist = INFINITY;
    best_cluster = -1;

    for (int k_i = 0; k_i < k; k_i++)
    {
      cur_dist = calc_distances(X[X_i], cluster_centers[k_i], d);
      if (cur_dist < best_dist)
      {
        best_dist = cur_dist;
        best_cluster = k_i;
      }
    }

    if (cluster_assignments[X_i] != best_cluster)
    {
      delta++;
    }

    cluster_assignments[X_i] = best_cluster;
  }

  return delta;
}

void calc_cluster_centers(int k, const float **X, int n, int d,
                          const int *cluster_assignments, float **cluster_centers)
{
  float **new_cluster_centers = NULL;
  if (!(new_cluster_centers = (float**)malloc(k*sizeof(float*))))
  {
    throw;
  }
  for (int k_i = 0; k_i < k; k_i++)
  {
    if (!(new_cluster_centers[k_i] = (float*)calloc(d,sizeof(float))))
    {
      throw;
    }
  }

  int *counts = (int*)calloc(k,sizeof(int));
  int cluster;
  for (int X_i = 0; X_i < n; X_i++)
  {
    cluster = cluster_assignments[X_i];

    counts[cluster]++;
    increment(new_cluster_centers[cluster], X[X_i], d);
  }

  for (int k_i = 0; k_i < k; k_i++)
  {
    if (counts[k_i])
    {
      divide(new_cluster_centers[k_i], counts[k_i], d);
    }
  }

  for (int k_i = 0; k_i < k; k_i++)
  {
    if (counts[k_i])
    {
      memcpy(cluster_centers[k_i], new_cluster_centers[k_i], d*sizeof(float));
    }
  }

  free_cluster_centers(k, new_cluster_centers, d);
  free(counts);
}

void increment(float* target, const float* value, int d)
{
  for (int d_i = 0; d_i < d; d_i++)
  {
    target[d_i] += value[d_i];
  }
}

void divide(float* target, float value, int d)
{
  for (int d_i = 0; d_i < d; d_i++)
  {
    target[d_i] = target[d_i] / value;
  }
}

float calc_distances(const float* p1, const float* p2, int d)
{
  float dist_sum = 0;
  float dist = 0;

  for (int d_i = 0; d_i < d; d_i++)
  {
    dist = p1[d_i] - p2[d_i];
    dist_sum += dist * dist;
  }

  return sqrt(dist_sum);
}

void free_cluster_centers(int k, float **cluster_centers, int d)
{
  for (int k_i = 0; k_i < k; k_i++)
  {
    free(cluster_centers[k_i]);
  }
  free(cluster_centers);
}
